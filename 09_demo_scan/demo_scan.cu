
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>

/*
Somethings so confuse me, why i can't get same correct result every time.
*/

/*
These two kernel could be used on large array, but slow
Best advice: use __syncthreads() before you want to use different index
*/
__global__ void hillis_steele_scan_forward(float * d_out, float * d_in, const int array_size){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	d_out[idx] = d_in[idx];	
	for(int step = 1; step < array_size; step *= 2){		
		if(idx + step >= array_size) return;
		__syncthreads();
		float in1 = d_out[idx];
		__syncthreads();
		d_out[idx + step] += in1;
	}
}

__global__ void hillis_steele_scan_backward(float * d_out, float * d_in){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	d_out[idx] = d_in[idx];
	for(int step = 1; step <= idx; step *= 2){		
		if(idx - step < 0) return;
		__syncthreads();
		float in1 = d_out[idx - step];
		__syncthreads();
		d_out[idx] += in1;
	}
}

/*
These two kernel could be used on small array, but fast
*/
__global__ void shared_hillis_steele_scan_forward(float *d_out, float *d_in, const int array_size) {
	extern __shared__ float sdata[];
	int idx = threadIdx.x;
	if(idx < array_size) {
		sdata[idx] = d_in[idx];
	} else {
		return;
	}	// the code below performs iterative scan on XY
	for(int step = 1; step < array_size; step *= 2){		
		if(idx + step >= array_size) return;
		__syncthreads();
		float in1 = sdata[idx];
		__syncthreads();
		sdata[idx + step] += in1;
	}
	d_out[idx] = sdata[idx];
}

__global__ void shared_hillis_steele_scan_backward(float * d_out, float * d_in, const int array_size){
	extern __shared__ float sdata[];
	int idx = threadIdx.x;
	if(idx < array_size) {
		sdata[idx] = d_in[idx];
	} else {
		return;
	}
	sdata[idx] = d_in[idx];
	for(int step = 1; step <= idx; step *= 2){		
		if(idx - step < 0) return;
		__syncthreads();
		float in1 = sdata[idx - step];
		__syncthreads();
		sdata[idx] += in1;
	}
	d_out[idx] = sdata[idx];
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 1025;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	const int maxThreadPerBlock = 512;
	const int numBlock = ARRAY_SIZE / maxThreadPerBlock + 1;
	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	//hillis_steele_scan_forward<<<numBlock, maxThreadPerBlock>>>(d_out, d_in, ARRAY_SIZE);
	//hillis_steele_scan_backward<<<numBlock, maxThreadPerBlock>>>(d_out, d_in);
	//shared_hillis_steele_scan_forward<<<numBlock, maxThreadPerBlock, maxThreadPerBlock  * sizeof(float)>>>(d_out, d_in, ARRAY_SIZE);
	shared_hillis_steele_scan_backward<<<numBlock, maxThreadPerBlock, maxThreadPerBlock  * sizeof(float)>>>(d_out, d_in, ARRAY_SIZE);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
